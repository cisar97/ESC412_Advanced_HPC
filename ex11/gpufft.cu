#include "gpufft.h"

#define USE_PLAN_MANY 1

static void CUDA_Abort(hipError_t rc, const char *fname, const char *file, int line) {
    fprintf(stderr,"%s error %d in %s(%d)\n%s\n", fname, rc, file, line, hipGetErrorString(rc));
    exit(1);
}
static void CUDA_Abort(hipfftResult rc, const char *fname, const char *file, int line) {
    fprintf(stderr,"%s error %d in %s(%d)\n", fname, rc, file, line);
    exit(1);
}
#define CUDA_CHECK(f,a) {auto rc = (f)a; if (rc!=0) CUDA_Abort(rc,#f,__FILE__,__LINE__);}

// Create a plan to do a 2D transform for the given grid (in-place)
hipfftHandle gpu_make_plan_2D(int nGrid) {
    hipfftHandle plan;
#if USE_PLAN_MANY
    int n[] = {nGrid,nGrid};       // 2D FFT of length NxN
    int inembed[] = {nGrid,2*(nGrid/2+1)};
    int onembed[] = {nGrid,nGrid/2+1};
    int howmany = 1;
    int odist = onembed[0] * onembed[1]; // Output distance is in "complex"
    int idist = 2*odist;   // Input distance is in "real"
    int istride = 1;       // Elements of each FFT are adjacent
    int ostride = 1;
    CUDA_CHECK(hipfftPlanMany,(&plan,sizeof(n)/sizeof(n[0]), n,
                    inembed,istride,idist,
                    onembed,ostride,odist,
                    HIPFFT_R2C,howmany));
#else
    CUDA_CHECK(hipfftPlan2d,(&plan,nGrid,nGrid,HIPFFT_R2C));
#endif
    return plan;
}

void gpu_fft_2D_R2C(blitz::Array<float,2> &grid,void *slab,hipfftHandle plan) {
    auto data_size = sizeof(hipfftComplex)*grid.rows()*(grid.cols()/2+1);
    CUDA_CHECK(hipMemcpy,(slab, grid.dataFirst(), data_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipfftExecR2C,(plan,reinterpret_cast<hipfftReal*>(slab),reinterpret_cast<hipfftComplex*>(slab)));
    CUDA_CHECK(hipMemcpy,(grid.dataFirst(), slab, data_size, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipDeviceSynchronize,());
}

void *gpu_allocate_slab(size_t nGrid) {
    void *cuda_slab;
    auto slab_size = sizeof(hipfftComplex)*nGrid*(nGrid/2+1);
    CUDA_CHECK(hipMalloc,((void**)&cuda_slab, slab_size));
    return cuda_slab;
}

