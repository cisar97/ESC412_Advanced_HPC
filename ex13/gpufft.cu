#include "gpufft.h"
#include "delta.h"
#include "cudacheck.h"

#define USE_PLAN_MANY 1

std::tuple<hipfftHandle,size_t> gpu_make_plan_1D(int nGrid) {
    hipfftHandle plan;
    int n[] = {nGrid};       // 2D FFT of length NxN
    int inembed[] = {nGrid};
    int onembed[] = {nGrid};
    int howmany = nGrid/2 + 1;	// There are this many pencils
    int odist = 1; 		// 1D pencils start next to each other
    int idist = 1;   		// 
    int istride = howmany;     	// Elements of each FFT are after each group
    int ostride = howmany;
    size_t workSize;
    CUDA_CHECK(hipfftCreate,(&plan));
    CUDA_CHECK(hipfftSetAutoAllocation,(plan,0));
    CUDA_CHECK(hipfftMakePlanMany,(plan,sizeof(n)/sizeof(n[0]), n,
                    inembed,istride,idist,
                    onembed,ostride,odist,
                    HIPFFT_C2C,howmany,&workSize));
    return {plan,workSize};
}

// Create a plan to do a 2D transform for the given grid (in-place)
std::tuple<hipfftHandle,size_t> gpu_make_plan_2D(int nGrid) {
    hipfftHandle plan;
#if USE_PLAN_MANY
    int n[] = {nGrid,nGrid};       // 2D FFT of length NxN
    int inembed[] = {nGrid,2*(nGrid/2+1)};
    int onembed[] = {nGrid,nGrid/2+1};
    int howmany = 1;
    int odist = onembed[0] * onembed[1]; // Output distance is in "complex"
    int idist = 2*odist;   // Input distance is in "real"
    int istride = 1;       // Elements of each FFT are adjacent
    int ostride = 1;
    size_t workSize;
    CUDA_CHECK(hipfftCreate,(&plan));
    CUDA_CHECK(hipfftSetAutoAllocation,(plan,0));
    CUDA_CHECK(hipfftMakePlanMany,(plan,sizeof(n)/sizeof(n[0]), n,
                    inembed,istride,idist,
                    onembed,ostride,odist,
                    HIPFFT_R2C,howmany,&workSize));
#else
    CUDA_CHECK(hipfftPlan2d,(&plan,nGrid,nGrid,HIPFFT_R2C));
#endif
    return {plan,workSize};
}

void gpu_fft_2D_R2C(blitz::Array<float,2> &grid,void *slab,hipfftHandle plan,hipStream_t stream,void *work,int nGrid,float diRhoBar) {
    auto fslab = static_cast<float*>(slab);
    auto data_size = sizeof(hipfftComplex)*grid.rows()*(grid.cols()/2+1);
    CUDA_CHECK(hipMemcpyAsync,(slab, grid.dataFirst(), data_size, hipMemcpyHostToDevice,stream));
    compute_delta(fslab,nGrid,diRhoBar,stream);
    CUDA_CHECK(hipfftSetStream,(plan,stream));
    CUDA_CHECK(hipfftSetWorkArea,(plan,work));
    CUDA_CHECK(hipfftExecR2C,(plan,reinterpret_cast<hipfftReal*>(slab),reinterpret_cast<hipfftComplex*>(slab)));
    CUDA_CHECK(hipMemcpyAsync,(grid.dataFirst(), slab, data_size, hipMemcpyDeviceToHost,stream));
}

void gpu_fft_1D_C2C(blitz::Array<std::complex<float>,2> &grid,void *slab,hipfftHandle plan,hipStream_t stream,void *work) {
    auto data_size = sizeof(hipfftComplex)*grid.rows()*(grid.cols());
    CUDA_CHECK(hipMemcpyAsync,(slab, grid.dataFirst(), data_size, hipMemcpyHostToDevice,stream));
    CUDA_CHECK(hipfftSetStream,(plan,stream));
    CUDA_CHECK(hipfftSetWorkArea,(plan,work));
    CUDA_CHECK(hipfftExecC2C,(plan,reinterpret_cast<hipfftComplex*>(slab),reinterpret_cast<hipfftComplex*>(slab),HIPFFT_FORWARD));
    CUDA_CHECK(hipMemcpyAsync,(grid.dataFirst(), slab, data_size, hipMemcpyDeviceToHost,stream));
}

void *gpu_allocate_slab(size_t nGrid) {
    void *cuda_slab;
    auto slab_size = sizeof(hipfftComplex)*nGrid*(nGrid/2+1);
    CUDA_CHECK(hipMalloc,((void**)&cuda_slab, slab_size));
    return cuda_slab;
}

void *gpu_allocate(size_t n) {
    void *data;
    CUDA_CHECK(hipMalloc,((void**)&data, n));
    return data;
}

